#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "gputimer.h"
#include "lodepng.h"
#include "wm.h" // for the 3x3 float "w" matrix

#define CLAMP_0_255(x) ((x) < 0 ? 0 : ((x) > 255 ? 255 : (x)))

// Declare a constant-memory version of the 3x3 filter "w"
__constant__ float d_w[3][3];

/**
 * @brief Device kernel that applies the 3x3 filter (in d_w) to each pixel (except borders).
 *        Expects an RGBA 8-bit input.
 */
__global__
void gpuConvolveKernel(const unsigned char* d_in,
                       unsigned char*       d_out,
                       unsigned width,
                       unsigned height)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // global thread ID

    // Only process the "inner" (width-2)*(height-2) region
    int totalInner = (width - 2) * (height - 2);
    if(gid >= totalInner) return;

    // Determine (row,col) in the output region
    int outRow = gid / (width - 2);
    int outCol = gid % (width - 2);

    // The corresponding input pixel is +1 offset
    int inRow = outRow + 1;
    int inCol = outCol + 1;

    float rVal = 0.f;
    float gVal = 0.f;
    float bVal = 0.f;

    // 3×3 filter accumulation
    for(int ii = 0; ii < 3; ii++)
    {
        for(int jj = 0; jj < 3; jj++)
        {
            int rr = inRow + (ii - 1);
            int cc = inCol + (jj - 1);

            int inIdx = (rr * width + cc) * 4;

            // Use the d_w constant array, not a host variable
            rVal += d_in[inIdx + 0] * d_w[ii][jj];
            gVal += d_in[inIdx + 1] * d_w[ii][jj];
            bVal += d_in[inIdx + 2] * d_w[ii][jj];
        }
    }

    // Write to the corresponding (outRow, outCol) in d_out
    int outIdx = (outRow * (width - 2) + outCol) * 4;

    d_out[outIdx + 0] = (unsigned char)CLAMP_0_255((int)rVal);
    d_out[outIdx + 1] = (unsigned char)CLAMP_0_255((int)gVal);
    d_out[outIdx + 2] = (unsigned char)CLAMP_0_255((int)bVal);

    // Preserve alpha from the center pixel
    int centerIdx = (inRow * width + inCol) * 4 + 3;
    d_out[outIdx + 3] = d_in[centerIdx];
}

/**
 * @brief Host function to manage the CUDA memory allocations and kernel launch.
 */
static void gpuConvolve(const unsigned char* h_in,
                        unsigned char*       h_out,
                        unsigned width,
                        unsigned height,
                        int threadsPerBlock)
{
    size_t inBytes  = (size_t)width * height * 4;    // RGBA input
    size_t outBytes = (size_t)(width - 2) * (height - 2) * 4;

    unsigned char *d_in = nullptr, *d_out = nullptr;
    hipMalloc(&d_in,  inBytes);
    hipMalloc(&d_out, outBytes);

    hipMemcpy(d_in, h_in, inBytes, hipMemcpyHostToDevice);

    // Configure kernel
    int totalInner = (width - 2) * (height - 2);
    int blockSize  = threadsPerBlock;
    int gridSize   = (totalInner + blockSize - 1) / blockSize;

    // Launch kernel
    gpuConvolveKernel<<<gridSize, blockSize>>>(d_in, d_out, width, height);
    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(h_out, d_out, outBytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

static void handleCudaError(hipError_t err, const char* msg)
{
    if(err != hipSuccess)
    {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char* argv[])
{
    if(argc != 4)
    {
        printf("Usage: %s <input.png> <output.png> <#threads>\n", argv[0]);
        return 1;
    }
    const char* inputFile  = argv[1];
    const char* outputFile = argv[2];
    int threads = atoi(argv[3]);
    if(threads <= 0)
    {
        printf("Error: #threads must be > 0\n");
        return 1;
    }

    // We'll copy "w" from wm.h into a host array and then to device constant memory
    float host_w[3][3] = {
        { w[0][0], w[0][1], w[0][2] },
        { w[1][0], w[1][1], w[1][2] },
        { w[2][0], w[2][1], w[2][2] }
    };
    // Copy to device constant memory
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_w), host_w, 9 * sizeof(float));
    handleCudaError(err, "hipMemcpyToSymbol failed for d_w");

    // Decode 32-bit RGBA
    unsigned width = 0, height = 0;
    unsigned char* imageData = nullptr;
    unsigned error = lodepng_decode32_file(&imageData, &width, &height, inputFile);
    if(error)
    {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        return 1;
    }

    if(width < 3 || height < 3)
    {
        printf("Error: image too small for 3x3 convolution.\n");
        free(imageData);
        return 1;
    }

    unsigned outW = width - 2;
    unsigned outH = height - 2;
    size_t outSize = (size_t)outW * outH * 4;

    unsigned char* outData = (unsigned char*)malloc(outSize);
    if(!outData)
    {
        printf("Error: could not allocate outData\n");
        free(imageData);
        return 1;
    }

    // Time the GPU convolve
    GpuTimer timer;
    timer.Start();
    gpuConvolve(imageData, outData, width, height, threads);
    timer.Stop();

    printf("GPU Convolution Time (%d threads): %.3f ms\n", threads, timer.Elapsed());

    // Encode as 32-bit RGBA
    error = lodepng_encode32_file(outputFile, outData, outW, outH);
    if(error)
    {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        free(imageData);
        free(outData);
        return 1;
    }

    printf("Output written to: %s\n", outputFile);

    free(imageData);
    free(outData);
    return 0;
}
