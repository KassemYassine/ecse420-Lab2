#include "hip/hip_runtime.h"
#include "common.h"
#include "gputimer.h"

/** @brief The size of the grid. Its square cannot exceed the maximum number of threads (1024). */
#define N 4
#if N*N > 1024
    #error "N*N must be less than 1024"
#endif

/**
 * @brief Returns the index of the element at (i, j) in the grid of size N. 
 */
#define idx(gridIndex, row, col) (((gridIndex) * N * N) + ((row) * N) + (col))


/**
 * @brief Simulates a drum sound via the finite element method.
 * 
 * @param iterations - The number of iterations to simulate
 * @param u - The grids containing the two previous states and the current state (to be computed).
 */
__global__ void simulate(int iterations, float* u) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    for (int iter = 0; iter < iterations; iter++) {
        // Compute the current state (interior elements)
        if (i > 0 && i < N-1 && j > 0 && j < N-1) {
            float newValue =
                u[idx(1,i-1,j)] + 
                u[idx(1,i+1,j)] + 
                u[idx(1,i,j-1)] + 
                u[idx(1,i,j+1)] - 
                4*u[idx(1,i,j)];
            newValue *= RHO;
            newValue += 2.0f*u[idx(1,i,j)];
            newValue -= (1-ETA)*u[idx(2,i,j)];
            u[idx(0,i,j)] = newValue / (1+ETA);
        }
        __syncthreads();

        // Compute the current state (boundary elements)
        if (i == 0 && j > 0 && j < N-1) {
            u[idx(0,0,j)] = G * u[idx(0,1,j)];
        }
        else if (i == N-1 && j > 0 && j < N-1) {
            u[idx(0,N-1,j)] = G * u[idx(0,N-2,j)];
        }
        else if (j == 0 && i > 0 && i < N-1) {
            u[idx(0,i,0)] = G * u[idx(0,i,1)];
        }
        else if (j == N-1 && i > 0 && i < N-1) {
            u[idx(0,i,N-1)] = G * u[idx(0,i,N-2)];
        }
        __syncthreads();

        // Compute the current state (corners)
        if (i == 0 && j == 0) 
            u[idx(0,0,0)] = G * u[idx(0,1,0)];
        else if (i == N-1 && j == 0)
            u[idx(0,N-1,0)] = G * u[idx(0,N-2,0)];
        else if (i == 0 && j == N-1)
            u[idx(0,0,N-1)] = G * u[idx(0,0,N-2)];
        else if (i == N-1 && j == N-1)
            u[idx(0,N-1,N-1)] = G * u[idx(0,N-1,N-2)];
        __syncthreads();

        // Shift the grids
        u[idx(2,i,j)] = u[idx(1,i,j)];
        u[idx(1,i,j)] = u[idx(0,i,j)];

        // Print the current state
        if (i == N/2 && j == N/2) {
            printf("(%d,%d): %f\n", i, j, u[idx(0,i,j)]);
        }
        __syncthreads();
    }
}

int main(int argc, char **argv) {
    // Validate arguments
    if (argc != 2) {
        printf("Usage: %s <number of iterations>\n", argv[0]);
        return 1;
    }

    // Get the number of iterations from the command line
    int iterations = atoi(argv[1]);

    hipError_t cudaStatus;
    GpuTimer timer;

    // Allocate memory for the grids
    float* u = (float*)malloc(3 * N * N * sizeof(float));
    
    // Initialize the grids with zeros and set the initial condition
    for (int i = 0; i < 3 * N * N; i++) {
        u[i] = 0.0f;
    }
    u[idx(1,N/2,N/2)] = 1.0f;

    // Allocate memory for the grids on the device
    float* d_u = NULL;
    cudaStatus = hipMalloc(&d_u, 3 * N * N * sizeof(float));
    checkCudaError(cudaStatus, "Failed to allocate memory on the device", Error);

    // Copy the grids to the device
    cudaStatus = hipMemcpy(d_u, u, 3 * N * N * sizeof(float), hipMemcpyHostToDevice);
    checkCudaError(cudaStatus, "Failed to copy memory to the device", Error);

    // Simulate the drum sound
    dim3 threadsPerBlock(N, N);
    timer.Start();
    simulate<<<1, threadsPerBlock>>>(iterations, d_u);
    cudaStatus = hipGetLastError();
    checkCudaError(cudaStatus, "Failed to launch kernel", Error);
    cudaStatus = hipDeviceSynchronize();
    checkCudaError(cudaStatus, "Failed to synchronize device", Error);
    timer.Stop();

    // Copy the grids from the device
    cudaStatus = hipMemcpy(u, d_u, 3 * N * N * sizeof(float), hipMemcpyDeviceToHost);
    checkCudaError(cudaStatus, "Failed to copy memory from the device", Error);

    // Print the elapsed time
    printf("Elapsed time: %f ms\n", timer.Elapsed());

Error:
    // Free memory
    free(u);
    hipFree(d_u);
    
    return 0;
}